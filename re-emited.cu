#include "hip/hip_runtime.h"
/* 
# Made by:
# Sergio Mendoza <sergio@mendozza.org>
# Milton Santibañez <msantibanez@astro.unam.mx>
# Gustavo Magallanes-Guijon <gustavo.magallanes.guijon@ciencias.unam.mx>
# Instituto de Astronomia UNAM
# Ciudad Universitaria
# Ciudad de Mexico
# Mexico
# Fri 21 Oct 2020 05:40:36 PM UTC
*/

//includes
#include "content/Headers.h"

// defines
#define COLUMNS 3 
#define LADO 400
#define ROWS LADO*LADO

#define MATRIX COLUMNS*ROWS

//Grid
#define THREADS 128 
#define BLOCKS (int)ceil(ROWS/THREADS) + 1


/*datafile to be open for write*/
FILE *datafilewrite;

/*datafile to be open for read*/
FILE *datafileread;

//////////////////////////////////////////////////////////////////////////////
////////////////////////  K E R N E L  ///////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
__global__ 
void remited(int k, 
	     int steps, 
	     double *dev_vector_emisor, 
	     double *dev_vector_receptor, 
	     double d_min, 
	     double l_min, 
	     double d_max, 
	     double l_max)
{

    double x_grid, y_grid, z_grid,  diff_y, diff_z;
  
    double Px, Py, Pz, Tau_x, Tau_y, Tau_z;
    double r_evol, theta_evol, phi_evol, r_pre, theta_pre, phi_pre, e_r, e_theta, e_phi, e_r_pre, e_theta_pre, e_phi_pre, dl;
    double distancia, r_orbita, dt, theta_0;
    int sentinel, fila;
  
    double epsilon = 0.1;
  
    sentinel = 0;
    distancia = 1000.0;
    theta_0 = 0.5*M_PI;
  
    dl = 0.01;
    r_orbita = 100.0;
    dt = M_PI/steps;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    //points of the grid which will be used like initial points in the evolution
    Px = r_orbita*cosf((0.5*M_PI)+k*dt);
    Py = dev_vector_emisor[(idx)*COLUMNS + 0] + r_orbita*sinf((0.5*M_PI)+k*dt);
    Pz = dev_vector_emisor[(idx)*COLUMNS + 1];

    //we obtain the initial values of Tau
    Tau_x = sinf(theta_0) * cosf(0.0);
    Tau_y = sinf(theta_0) * sinf(0.0);
    Tau_z = cosf(theta_0);

    //There are the initialization of the coordenates and the vector e for de leap frog algoritm
    r_evol = sqrt( Px*Px + Py*Py + Pz*Pz );
    if(Pz > 0.0)theta_evol = atanf(sqrt( Px*Px + Py*Py )/Pz);
    if(Pz == 0.0)theta_evol = 0.5*M_PI;
    if(Pz < 0.0)theta_evol = M_PI + atanf(sqrt( Px*Px + Py*Py )/Pz);
    if(Px > 0.0 && Py >= 0.0)phi_evol = atanf(Py/Px);
    if(Px > 0.0 && Py < 0.0)phi_evol = 2.0*M_PI + atanf(Py/Px);
    if(Px == 0.0)phi_evol = copysign(0.5*M_PI,Py);
    if(Px < 0.0)phi_evol = M_PI + atanf(Py/Px);

    e_r = Tau_x*sinf(theta_evol)*cosf(phi_evol) + Tau_y*sinf(theta_evol)*sinf(phi_evol) + Tau_z*cosf(theta_evol);
    e_theta = Tau_x*cosf(theta_evol)*cosf(phi_evol)/r_evol + Tau_y*cosf(theta_evol)*sinf(phi_evol)/r_evol - Tau_z*sinf(theta_evol)/r_evol;
    e_phi = -1.0*Tau_x*sinf(phi_evol)/(r_evol*sinf(theta_evol)) + Tau_y*cosf(phi_evol)/(r_evol*sinf(theta_evol));

    #pragma unroll 
    for (;;){
        //We rename the variables for the next calculus
        r_pre = r_evol; 
        theta_pre = theta_evol;
        phi_pre = phi_evol;
  
        e_r_pre = e_r;
        e_theta_pre = e_theta;
        e_phi_pre = e_phi;
  
        //We evolve the coordenates and the components of e, with the variables of the past step
        r_evol = r_pre + e_r_pre*dl;
        theta_evol = theta_pre + e_theta_pre*dl;
        phi_evol = phi_pre + e_phi_pre*dl;
  
        e_r = e_r_pre + dl*de_r(r_pre,theta_pre,phi_pre,e_r_pre,e_theta_pre,e_phi_pre);
        e_theta = e_theta_pre + dl*de_theta(r_pre,theta_pre,phi_pre,e_r_pre,e_theta_pre,e_phi_pre);
        e_phi = e_phi_pre + dl*de_phi(r_pre,theta_pre,phi_pre,e_r_pre,e_theta_pre,e_phi_pre); 
  
        //We write the next point in cartesian coordenates
        Px = r_evol*sinf(theta_evol)*cosf(phi_evol);
        Py = r_evol*sinf(theta_evol)*sinf(phi_evol);
        Pz = r_evol*cosf(theta_evol);
  
        x_grid = Px - r_orbita*cosf((0.5*M_PI)+k*dt);
        y_grid = Py - r_orbita*sinf((0.5*M_PI)+k*dt);
        z_grid = Pz;
  

	if( 0.5*distancia - epsilon <= x_grid && x_grid <= 0.5*distancia + epsilon
  		       && d_min <= y_grid && y_grid <= d_max
  		       && l_min <= z_grid && z_grid <= l_max && sentinel==0 ){
             sentinel = 1;
             diff_y = 1.0*LADO;
             diff_z = 1.0*LADO;

             fila = 0;
             #pragma unroll
	     for(int m = 0 ; m < ROWS ; m++){
          	  if(fabsf(dev_vector_receptor[m*COLUMNS + 0] - y_grid) <= diff_y && 
		     fabsf(dev_vector_receptor[m*COLUMNS + 1] - z_grid) <= diff_z ){
              	          diff_y = fabsf(dev_vector_receptor[m*COLUMNS + 0] - y_grid);
  		          diff_z = fabsf(dev_vector_receptor[m*COLUMNS + 1] - z_grid);
        	          fila = m;
            	  }//END IF
             }//END FOR

	atomicAdd(&dev_vector_receptor[fila*COLUMNS + 2], dev_vector_emisor[idx*COLUMNS + 2]);

__syncthreads();
	}//END IF

    if(!( r_evol<=(1.1*distancia) && r_evol>1.1 && sentinel==0)) break;
    }//END FOR 
    sentinel = 0;   
}

//////////////////////////////////////////////////////////////////////////////
////////////////////////////////  D E V I C E  //////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

void onDevice(  double* hst_vector_emisor, 
		double* hst_vector_receptor, 
		double d_min, 
		double l_min, 
		double d_max, 
		double l_max  )
{
 // configuration Grids and Theads
 // dim3 GridBlocks(50,100);
 // dim3 ThreadsBlocks(8,4);


    //INIT KERNEL LOOP
    int steps = 180;

    //NAME OF FILE
    char nombre[20];

    // start timer
    GpuTimer timer;
    timer.Start();

    #pragma unroll steps
    for(int k=0; k<360; k++){

        double *dev_vector_emisor, *dev_vector_receptor;

       //hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

        // memory in the device
        HANDLER_ERROR_ERR(hipMalloc((void**)&dev_vector_emisor, MATRIX * sizeof(double)));
        HANDLER_ERROR_ERR(hipMalloc((void**)&dev_vector_receptor, MATRIX * sizeof(double)));
        
        // copy the data to the device
        HANDLER_ERROR_ERR(hipMemcpy(dev_vector_emisor, hst_vector_emisor, MATRIX * sizeof(double), 
				hipMemcpyHostToDevice));

	for(int i = 0; i < ROWS; i++){
	   hst_vector_receptor[i*3 + 2] = 0.0;
	  }

        // copy the data to the device
        HANDLER_ERROR_ERR(hipMemcpy(dev_vector_receptor, hst_vector_receptor, MATRIX * sizeof(double), 
				hipMemcpyHostToDevice));

        sprintf(nombre,"second_display_image%d.dat",k);
        
        datafilewrite = fopen ( nombre , "w" );

        ///////////////////////////////////////////////  K E R N E L  /////////////////////////////////////////////////////
        //remited<<<GridBlocks,ThreadsBlocks>>>(k, steps, dev_vector_emisor, dev_vector_receptor, d_min, l_min, d_max, l_max);
        remited<<<BLOCKS,THREADS>>>(k, steps, dev_vector_emisor, dev_vector_receptor, d_min, l_min, d_max, l_max);
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        HANDLER_ERROR_MSG("Kernel Panic!!!");
        
        // data from the  host to device
        HANDLER_ERROR_ERR(hipMemcpy(hst_vector_receptor, dev_vector_receptor, MATRIX  * sizeof(double), 
				hipMemcpyDeviceToHost));

	#pragma unroll ROWS
        for (int i = 0; i<ROWS; i++){
            if(hst_vector_receptor[(i)*COLUMNS + 0] != 0 && hst_vector_receptor[(i)*COLUMNS + 1] != 0){
          		fprintf(datafilewrite, "%lE \t %lE \t %lE \n", hst_vector_receptor[(i)*COLUMNS + 0]* 5.029643510174233, \
           						               hst_vector_receptor[(i)*COLUMNS + 1]* 5.029643510174233, \
           							       hst_vector_receptor[(i)*COLUMNS + 2]);
            }
        }

        fclose(datafilewrite);

        // liberamos memoria del device
        HANDLER_ERROR_ERR(hipFree( dev_vector_receptor ));
        HANDLER_ERROR_ERR(hipFree( dev_vector_emisor ));
    } //FOR 
  
    timer.Stop();

    // print time
//    printf("Time :  %f ms\n", timer.Elapsed());
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////  H O S T ////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
void onHost()
{
  
    double y, z, intensity;
  
    double d_min, l_min, d_max, l_max;
  
    d_min = 0.0;
    d_max = 0.0;
    l_min = 0.0;
    l_max = 0.0;
  
    // declaration of vectors
    double *hst_vector_emisor, *hst_vector_receptor;
    
    // memory in the  host
    hst_vector_emisor = (double*)malloc(MATRIX * sizeof(double));
    hst_vector_receptor = (double*)malloc(MATRIX * sizeof(double));
    
    // init the input file
    datafileread = fopen ( "emited_image.dat" , "r" );
  
    #pragma unroll ROWS
    for(int i = 0; i < ROWS; i++){
        fscanf(datafileread,"%lE\t%lE\t%lE\n", &y, &z, &intensity);
        hst_vector_emisor[(i)*COLUMNS + 0] = y / 5.029643510174233;
        hst_vector_emisor[(i)*COLUMNS + 1] = z / 5.029643510174233;
        hst_vector_emisor[(i)*COLUMNS + 2] = intensity;
        hst_vector_receptor[(i)*COLUMNS + 0] = hst_vector_emisor[(i)*COLUMNS + 0];
        hst_vector_receptor[(i)*COLUMNS + 1] = hst_vector_emisor[(i)*COLUMNS + 1];

        if(d_max < hst_vector_emisor[(i)*COLUMNS + 0]) d_max = hst_vector_emisor[(i)*COLUMNS + 0];
        if(d_min > hst_vector_emisor[(i)*COLUMNS + 0]) d_min = hst_vector_emisor[(i)*COLUMNS + 0];
        if(l_max < hst_vector_emisor[(i)*COLUMNS + 1]) l_max = hst_vector_emisor[(i)*COLUMNS + 1];
        if(l_min > hst_vector_emisor[(i)*COLUMNS + 1]) l_min = hst_vector_emisor[(i)*COLUMNS + 1];
    }

    fclose(datafileread);

    // start timer
    CpuTimer timer;
    timer.Start();
  
    ///////////////////////////////  D E V I C E  ////////////////////////////////
    onDevice(hst_vector_emisor, hst_vector_receptor,  d_min, l_min, d_max, l_max);
    //////////////////////////////////////////////////////////////////////////////
  
    // stop timer
    timer.Stop();
    // print time
    //printf("CPU Time :  %f ms\n", timer.Elapsed());
  
  
    //salida del programa
    free(hst_vector_receptor);
    free(hst_vector_emisor);
//    printf("-: successful execution :-\n");
}

//////////////////////////////////////////////////////////////////////////////
/////////////////////////  M A I N ///////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

int main() 
{
    onHost();
    return 0;
}

